#include "hip/hip_runtime.h"
#include "edsdca/util/math_utils.h"
#include <iostream>
#ifdef GPU

__global__
void vector_prod_gpu(double* x, double* y, double* res, long n) {
  //
  long i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    res[i] = x[i] * y[i];
  }
}

#define MAX_SIZE 2048
__global__
void vector_dot_gpu(double* x, double* y, double* res, long n) {
  __shared__ double temp[MAX_SIZE];
  temp[threadIdx.x] = x[threadIdx.x] * y[threadIdx.x];

  __syncthreads();

  if (0 == threadIdx.x) {
    double sum = 0;
    for (int i = 0; i < n; ++i) {
      sum += temp[i];
    }
    *res = sum;
  }
}


double NormSquared_gpu(const Eigen::VectorXd &x) {
  return VectorDotProd_gpu(x, x);
}


double VectorDotProd_gpu(const Eigen::VectorXd &x, const Eigen::VectorXd &y) {
  int block_size, grid_size, min_grid_size;
  hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                                     vector_dot_gpu, 0, 0);
  grid_size = (x.size() + block_size - 1) / block_size;
  grid_size = std::max(grid_size, min_grid_size);

  double *d_x = edsdca::memory::MemSync::PushToGpuX(x);
  double *d_y = edsdca::memory::MemSync::PushToGpuY(y);

  // Call the kernel
  vector_prod_gpu<<<grid_size, block_size>>>(edsdca::memory::MemSync::dx_,
         edsdca::memory::MemSync::dy_,
         edsdca::memory::MemSync::res_, x.size());

  // Copy back from gpu
  Eigen::VectorXd result =
      edsdca::memory::MemSync::PullFromGpu(edsdca::memory::MemSync::res_, x.size());

  return result.sum();
}


Eigen::VectorXd VectorReduce_gpu(const std::vector<Eigen::VectorXd> &v) {
  Eigen::VectorXd accumulator = Eigen::VectorXd::Zero(v.front().size());
  for (const Eigen::VectorXd& x : v) {
      accumulator += x;
  }
  return accumulator;
}

#endif // GPU
