
#include <hip/hip_runtime.h>
#ifdef GPU

#include "edsdca/memory/memsync.h"


namespace edsdca {
namespace memory {

  /**
   * Puts data stored by @p v onto GPU memory
   *
   * @param v data to be transfered to GPU
   * @param size size of the required data buffer (in bytes)
   *
   * @return a pointer to the memory location on the GPU where the pushed data
   *        is stored
   */
  double* MemSync::PushToGpu(const Eigen::VectorXd& x) {
    int size = x.size();
    double *cv = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; ++i) {
        cv[i] = x(i);
    }
    double* d_v = MemSync::AllocateMemOnGpu(size);
    cudaMemcpy(d_v, cv, sizeof(double) * size, cudaMemcpyHostToDevice);
    free(cv);
    return d_v;
  }

  /**
   * Pulls data from the GPU to host
   *
   * @param d_v pointer to the data on GPU
   * @param size size of the data buffer required to store the data (in bytes)
   *
   * @return pointer to the location of the data on host memory
   */
  Eigen::VectorXd MemSync::PullFromGpu(double* d_v, long size) {
    double* v = (double*)malloc(sizeof(double) * size);
    cudaMemcpy(v, d_v, sizeof(double) * size, cudaMemcpyDeviceToHost);
    Eigen::VectorXd eig_v(size);
    for (long i = 0; i < size; ++i) {
        eig_v(i) = v[i];
    }
    return eig_v;
  }

  /**
   * Allocates memory on the GPU
   *
   * @param size size of requested memory in bytes
   *
   * @return returns a pointer to the allocated memory on GPU
   */
  double* MemSync::AllocateMemOnGpu(const long size) {
    double* d_v = (double*)malloc(sizeof(double) * size);
    cudaMalloc((void**)&d_v, sizeof(double) * size);
    return d_v;
  }

  double MemSync::PullValFromGpu(double* d_x) {
    double res;
    double* tmp = (double*)malloc(sizeof(double));
    cudaMemcpy(tmp, d_x, sizeof(double), cudaMemcpyDeviceToHost);
    res = *tmp;
    free(tmp);
    return res;
  }  
} // memory
} // edsdca

#endif // GPU
