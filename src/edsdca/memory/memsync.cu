
#include <hip/hip_runtime.h>
#ifdef GPU

#include "edsdca/memory/memsync.h"

namespace edsdca {
namespace memory {
    
  /**
   * Puts data from matrix @p X onto GPU memory
   *
   * @param X data to be transfered to GPU memory
   */
  double *MemSync::PushToGpuMatrix(const std::vector<Eigen::VectorXd> &X) {
    int rows = X.size();
    int cols = X.front().size();
    double *cx = (double*)malloc(sizeof(double) * rows * cols);
    for (int i = 0; i < rows; ++i) {
      for (int j = 0; j < cols; ++j) {
        cx[i * rows + j] = X[i](j);
      }
    }

    if (!memory_is_allocated_ || rows * cols != MemSync::matrix_size_) {
      SetMatrixMemoryAllocationSize(rows * cols);
      AllocateGlobalSharedMem();
    }
    cudaMemcpy(MemSync::dX_, cx, sizeof(double) * rows * cols,
            cudaMemcpyHostToDevice);
    free(cx);
    return dX_;
  }


  /**
   * Puts data stored by @p v onto GPU memory
   *
   * @param v data to be transfered to GPU
   * @param size size of the required data buffer (in bytes)
   *
   * @return a pointer to the memory location on the GPU where the pushed data
   *        is stored
   */
  double* MemSync::PushToGpuX(const Eigen::VectorXd& x) {
    int size = x.size();
    double *cv = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; ++i) {
        cv[i] = x(i);
    }

    // Make sure memory is allocated before proceeding
    if (!memory_is_allocated_ || size != MemSync::d_) {
        SetMemoryAllocationSize(size);
        AllocateGlobalSharedMem();
    }
    cudaMemcpy(MemSync::dx_, cv, sizeof(double) * size, cudaMemcpyHostToDevice);
    free(cv);
    return dx_;
  }
    
  /**
   * Puts data stored by @p v onto GPU memory
   *
   * @param v data to be transfered to GPU
   * @param size size of the required data buffer (in bytes)
   *
   * @return a pointer to the memory location on the GPU where the pushed data
   *        is stored
   */
  double* MemSync::PushToGpuY(const Eigen::VectorXd& x) {
    int size = x.size();
    double *cv = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; ++i) {
        cv[i] = x(i);
    }

    // Make sure memory is allocated before proceeding
    if (!memory_is_allocated_ || size != MemSync::d_) {
        SetMemoryAllocationSize(size);
        AllocateGlobalSharedMem();
    }
    cudaMemcpy(MemSync::dy_, cv, sizeof(double) * size, cudaMemcpyHostToDevice);
    free(cv);
    return dy_;
  }

  /**
   * Pulls data from the GPU to host
   *
   * @param d_v pointer to the data on GPU
   * @param size size of the data buffer required to store the data (in bytes)
   *
   * @return pointer to the location of the data on host memory
   */
  Eigen::VectorXd MemSync::PullFromGpu(double* d_v, long size) {
    double* v = (double*)malloc(sizeof(double) * size);
    cudaMemcpy(v, d_v, sizeof(double) * size, cudaMemcpyDeviceToHost);
    Eigen::VectorXd eig_v(size);
    for (long i = 0; i < size; ++i) {
        eig_v(i) = v[i];
    }
    return eig_v;
  }

  Eigen::MatrixXd MemSync::PullMatrixFromGpu() {
    int cols = d_;
    int rows = int(matrix_size_ / d_);
    double *tmpX = (double*)malloc(sizeof(double) * matrix_size_);
    cudaMemcpy(tmpX, dX_, sizeof(double) * matrix_size_,
            cudaMemcpyDeviceToHost);
    Eigen::MatrixXd result(rows, cols);
    for (int i = 0; i < rows; ++i) {
      for (int j = 0; j < cols; ++j) {
        result(i, j) = tmpX[i * rows + i];
      }
    }
    free(tmpX);
    return result;
  }

  Eigen::VectorXd MemSync::PullResFromGpu() {
    double *tmpR = (double*)malloc(sizeof(double) * d_);
    cudaMemcpy(tmpR, res_, sizeof(double) * d_, cudaMemcpyDeviceToHost);
    Eigen::VectorXd result(d_);
    for (long i = 0; i < d_; ++i) {
      result(i) = tmpR[i];
    }
    free(tmpR);
    return result;
  }

  /**
   * Allocates memory on the GPU
   *
   * @param size size of requested memory in bytes
   *
   * @return returns a pointer to the allocated memory on GPU
   */
  double* MemSync::AllocateMemOnGpu(const long size) {
    double* d_v = (double*)malloc(sizeof(double) * size);
    cudaMalloc((void**)&d_v, sizeof(double) * size);
    return d_v;
  }

  double MemSync::PullValFromGpu(double* d_x) {
    double res;
    double* tmp = (double*)malloc(sizeof(double));
    cudaMemcpy(tmp, d_x, sizeof(double), cudaMemcpyDeviceToHost);
    res = *tmp;
    free(tmp);
    return res;
  }

  void MemSync::AllocateGlobalSharedMem() {
    if (memory_is_allocated_) {
        cudaFree(dx_);
        cudaFree(dy_);
        cudaFree(res_);
        cudaFree(dX_);
    }
    cudaMalloc((double**)&dx_, d_ * sizeof(double));
    cudaMalloc((double**)&dy_, d_ * sizeof(double));
    cudaMalloc((double**)&res_, d_ * sizeof(double));
    cudaMalloc((double**)&dX_, matrix_size_ * sizeof(double));
    memory_is_allocated_ = true;
  }
} // memory
} // edsdca

#endif // GPU
